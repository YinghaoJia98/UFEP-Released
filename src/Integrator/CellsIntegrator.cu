#include "hip/hip_runtime.h"
#include <vrmapping/Integrator/CellsIntegrator.h>

namespace CellsIntegratorGPUNS
{
    __device__ __constant__ double ResolutionInDevice_;
    double ResolutionInHost_;

    // Function to set the resolution (must be a host function)
    void setResolution(double Resolution)
    {
        hipMemcpyToSymbol(HIP_SYMBOL(ResolutionInDevice_), &Resolution, sizeof(double));
        ResolutionInHost_ = Resolution;
        // HostPrintResolutionInDevice();
    }

    // Kernel to check the value on the device
    __global__ void printResolutionInDevice()
    {
        printf("ResolutionInDevice_ is %f\n", ResolutionInDevice_);
    }

    // __device__ function for calculating X index
    __device__ int CalculateXIndex(double x, double minX)
    {
        return static_cast<int>((x - minX) / ResolutionInDevice_);
    }

    // __device__ function for calculating Y index
    __device__ int CalculateYIndex(double y, double minY)
    {
        return static_cast<int>((y - minY) / ResolutionInDevice_);
    }

    // CUDA Kernel for integrating cells
    __global__ void integrateCellsKernel(double *OutputTraversability, double *OutputElevation, const PlannerCell *d_Cells, int numCells,
                                         int numRows, int numCols, double minX, double minY)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;

        if (idx < numCells)
        {
            // Access individual elements of PlannerCell (Eigen::Vector4d)
            double x = d_Cells[idx][0];
            double y = d_Cells[idx][1];
            // printf("x is %f.", x);
            // printf("y is %f.", y);
            double ElevationTem_ = d_Cells[idx][2];
            double TraversabilityTem_ = d_Cells[idx][3];

            // Calculate grid indices using the custom functions
            int row = CalculateYIndex(y, minY);
            int col = CalculateXIndex(x, minX);

            if (row >= 0 && row < numRows && col >= 0 && col < numCols)
            {
                OutputTraversability[row * numCols + col] = TraversabilityTem_;
                OutputElevation[row * numCols + col] = ElevationTem_;
            }
        }
    }

    // Function to integrate cells on the GPU
    void integrateCellsGPU(const std::vector<PlannerCell> &Cells1, const std::vector<PlannerCell> &Cells2,
                           double minX, double maxX, double minY, double maxY, std::vector<PlannerCell> &integratedCells)
    {
        // std::cout << "Max and Min X are" << maxX << " and " << minX << std::endl;
        // std::cout << "Max and Min Y are" << maxY << " and " << minY << std::endl;
        // std::cout << "ResolutionInDevice_ is " << ResolutionInDevice_ << std::endl;
        // std::cout << "ResolutionInHost_ is " << ResolutionInHost_ << std::endl;
        int numRows = static_cast<int>(std::ceil((maxY - minY) / ResolutionInHost_) + 1);
        int numCols = static_cast<int>(std::ceil((maxX - minX) / ResolutionInHost_) + 1);
        // std::cout << "numRows is " << numRows << std::endl;
        // std::cout << "numCols is " << numCols << std::endl;
        int numCells1 = Cells1.size();
        int numCells2 = Cells2.size();
        // std::cout << "numCells1 is " << numCells1 << std::endl;
        // std::cout << "numCells2 is " << numCells2 << std::endl;

        // Host and device memory allocation
        std::vector<double> outputTra(numRows * numCols, -1.0); // Default to -1 for uninitialized cells
        std::vector<double> outputEle(numRows * numCols, -1.0); // Default to -1 for uninitialized cells
        // std::cout << "Initially, the size of outputTra is " << outputTra.size() << std::endl;
        // std::cout << "Initially, the size of outputEle is " << outputEle.size() << std::endl;
        PlannerCell *Device_Cells1;
        PlannerCell *Device_Cells2;
        double *Device_OutputTraversability_;
        double *Device_OutputElevation_;

        hipMalloc(&Device_Cells1, numCells1 * sizeof(PlannerCell));
        hipMalloc(&Device_Cells2, numCells2 * sizeof(PlannerCell));
        hipMalloc(&Device_OutputTraversability_, numRows * numCols * sizeof(double));
        hipMalloc(&Device_OutputElevation_, numRows * numCols * sizeof(double));

        // Copy data to device
        hipMemcpy(Device_Cells1, Cells1.data(), numCells1 * sizeof(PlannerCell), hipMemcpyHostToDevice);
        hipMemcpy(Device_Cells2, Cells2.data(), numCells2 * sizeof(PlannerCell), hipMemcpyHostToDevice);
        hipMemcpy(Device_OutputTraversability_, outputTra.data(), numRows * numCols * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(Device_OutputElevation_, outputEle.data(), numRows * numCols * sizeof(double), hipMemcpyHostToDevice);

        // Kernel execution for Cells2 first
        int threadsPerBlock = 256;
        int blocksPerGrid = (numCells2 + threadsPerBlock - 1) / threadsPerBlock;
        integrateCellsKernel<<<blocksPerGrid, threadsPerBlock>>>(Device_OutputTraversability_, Device_OutputElevation_, Device_Cells2, numCells2, numRows, numCols, minX, minY);
        hipDeviceSynchronize();

        // Kernel execution for Cells1, which will overwrite conflicts
        blocksPerGrid = (numCells1 + threadsPerBlock - 1) / threadsPerBlock;
        integrateCellsKernel<<<blocksPerGrid, threadsPerBlock>>>(Device_OutputTraversability_, Device_OutputElevation_, Device_Cells1, numCells1, numRows, numCols, minX, minY);
        hipDeviceSynchronize();

        // Copy result back to host
        hipMemcpy(outputTra.data(), Device_OutputTraversability_, numRows * numCols * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(outputEle.data(), Device_OutputElevation_, numRows * numCols * sizeof(double), hipMemcpyDeviceToHost);

        // std::cout << "outputTra.size is " << outputTra.size() << std::endl;
        // std::cout << "outputEle.size is " << outputEle.size() << std::endl;
        // Transform d_output into std::vector<PlannerCell>
        for (int i = 0; i < numRows; ++i)
        {
            for (int j = 0; j < numCols; ++j)
            {
                double TraversabilityTem_ = outputTra[i * numCols + j];
                double ElevationTem_ = outputEle[i * numCols + j];
                if (TraversabilityTem_ != -1)
                { // Ignore uninitialized cells
                    // TODO Might need to be rewritten, the 0.5 is a small hot code.
                    double x = minX + (double)(j + 0.5) * ResolutionInHost_;
                    double y = minY + (double)(i + 0.5) * ResolutionInHost_;
                    integratedCells.emplace_back(x, y, ElevationTem_, TraversabilityTem_);
                }
            }
        }
        // std::cout << "integratedCells.size is " << integratedCells.size() << std::endl;

        // Cleanup
        hipFree(Device_Cells1);
        hipFree(Device_Cells2);
        hipFree(Device_OutputTraversability_);
        hipFree(Device_OutputElevation_);
    }

    void HostPrintResolutionInDevice()
    {
        // Launch a kernel to print the resolution from the device side
        printResolutionInDevice<<<1, 1>>>();
        hipDeviceSynchronize();
    }
} // namespace CellsIntegratorGPUNS
